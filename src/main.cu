
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <array>
#include <iostream>

#ifndef EXAMPLE
  #define EXMAPLE 1
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

const int N = 1 << 20;

__global__ void calculation_kernel(float *x, int n)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    x[i] = sqrt(pow(3.14159,i));
  }
}


//-----------------------------------------------------------------------------

template <int TASK_COUNT>
class TaskStreamGraph
{
  public:

    TaskStreamGraph(){
      for (int i = 0; i < TASK_COUNT; ++i){

	// Initialize arrays
	d_count[i] = 0;
	for (int j = 0; j < TASK_COUNT; ++j){
	  dep_matrix[i][j] = 0;
	}
	
	// Initialize streams and memory for cuda kernels. 
	hipStreamCreate(&streams[i]);
	hipMalloc(&data[i], N * sizeof(float));
	hipEventCreateWithFlags(&events[i], hipEventDisableTiming);
      }
    }

    // **********************************************************
    // Fills the dependency matrix with user defined task dependencies
    void dependentOn(int m, const std::vector<int> &n)
    {
      for (int i = 0; i < n.size(); ++i){
	if (dep_matrix[m][n[i]] != 1) {
	  dep_matrix[m][n[i]] = 1;
	  d_count[m]++;
	}
      }
    }


    // **********************************************************
    // Return a list of dependents for a given task
    std::vector<int> getDependents(int task){
      std::vector<int> results;
      for(int m = 0; m < TASK_COUNT; ++m){
	if (dep_matrix[m][task] == 1) results.push_back(m);
      }
      return results;
    }


    // **********************************************************
    // Generating an execution list so cuda can call the kernels and appropriate
    // events in the correct order from the host.
    void buildAndExecuteGraph()
    {
      std::vector<int> pass_list;

      auto temp_d_count = d_count;
      exec_list.clear();

      // Generate the execution list. this defines the order kernels should be called 
      // based on the dependency matrix.
      do{

	// For each pass over the tasks determine which tasks have no dependencies remaining.
	pass_list.clear();
	for (int m = 0; m < TASK_COUNT; ++m){
	  if (temp_d_count[m] == 0){
	    pass_list.push_back(m);
	  }
	}

	// For each new task in the pass list decrement their respective dependents d_count value.
	for (int i = 0; i < pass_list.size(); ++i){
	  int n = pass_list[i];
	  for (int m = 0; m < TASK_COUNT; ++m){
	    if (dep_matrix[m][n] == 1){
	      temp_d_count[m]--;
	    }
	  }
	  // Set task count to -1, indicating it has been pushed to the execution list.
	  temp_d_count[n] = -1;
	}

	// Append pass list to the execution list.
	exec_list.insert(exec_list.end(), pass_list.begin(), pass_list.end());

      }while(pass_list.size() > 0);


    // **********************************************************
      // Launch and print execution list
      for (int p : exec_list)
      {
	std::cout << p << " ";
	launch(p, getDependents(p));
      }
      std::cout << std::endl;
    }


    // **********************************************************
    // Print deppendency matrix 
    void print_dep_matrix(){
      for (int m = 0; m < TASK_COUNT; ++m){
	for (int n = 0; n < TASK_COUNT; ++n){
	  std::cout << dep_matrix[m][n] << " ";
	}
	std::cout << " - " << d_count[m] << std::endl;
      }
    }


  private:
    hipStream_t streams[TASK_COUNT];
    hipEvent_t events[TASK_COUNT];

    float *data[TASK_COUNT];
    int dep_matrix[TASK_COUNT][TASK_COUNT]; // [M][N] M is dependent on N
    std::array<int, TASK_COUNT> d_count;

    std::vector<int> exec_list;


    // **********************************************************
    // Launch demo kernel and define strema wait events baed on dependents.
    void launch(int task, std::vector<int> dependants){
      calculation_kernel<<<1, 64, 0, streams[task]>>>(data[task], N);
      gpuErrchk(  hipEventRecord(events[task], streams[task])  );

      for (int i = 0; i < dependants.size(); i++) {
	gpuErrchk(  hipStreamWaitEvent(streams[dependants[i]], events[task], 0)  );
      }
    }

};


int main()
{
#if EXAMPLE==1
  TaskStreamGraph<3> tsg;

  tsg.dependentOn(1, {0} );
  tsg.dependentOn(2, {0} );

#elif EXAMPLE==2
  TaskStreamGraph<8> tsg;

  tsg.dependentOn(2, {0,1});
  tsg.dependentOn(3, {0,2});
  tsg.dependentOn(4, {3});
  tsg.dependentOn(5, {2});
  tsg.dependentOn(6, {4,5});
  tsg.dependentOn(7, {4,5});

#else
  TaskStreamGraph<16> tsg;

  tsg.dependentOn(1, {0});
  tsg.dependentOn(2, {3});
  tsg.dependentOn(4, {0});
  tsg.dependentOn(5, {1,4});
  tsg.dependentOn(6, {5,2,7});
  tsg.dependentOn(7, {3});
  tsg.dependentOn(8, {12});
  tsg.dependentOn(9, {8,13});
  tsg.dependentOn(10, {9,6,11});
  tsg.dependentOn(11, {15});
  tsg.dependentOn(13, {12});
  tsg.dependentOn(14, {15});

#endif

  tsg.print_dep_matrix();
  tsg.buildAndExecuteGraph();

  hipDeviceReset();

  return 0;
}
